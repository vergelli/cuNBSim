#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include "body.cuh"
#include "cuda_utils.cuh"

// Función que gestiona la reserva de memoria para el array de cuerpos y los estados de hiprand
void bodyForceMalloc(int bytes, Body *p, Body *&p_device, hiprandState* &d_states, int nBodies) {

    printf("INFO - Allocating particles state on GPU\n");
    CHECK_CUDA_ERROR(hipMalloc((void**)&d_states, nBodies * sizeof(hiprandState)));

    printf("INFO - Allocating particles on GPU\n");
    CHECK_CUDA_ERROR(hipMalloc(&p_device, bytes));

    printf("INFO - starting memory migrate operation to GPU\n");
    CHECK_CUDA_ERROR(hipMemcpy(p_device, p, bytes, hipMemcpyHostToDevice));
}

void cudaFreeMemRoutines(Body *p_device, hiprandState *d_states, float *buf){

    printf("INFO - Freeing memory on GPU\n");
    CHECK_CUDA_ERROR(hipFree(d_states));
    CHECK_CUDA_ERROR(hipFree(p_device));
    printf("INFO - Freeing memory on CPU\n");
    free(buf);

}
