#include <hip/hip_runtime.h>
#include "body.cuh"
#include "cuda_utils.cuh"

void bodyForceMalloc(int bytes, Body *p, Body *&p_device) {
    CHECK_CUDA_ERROR(hipMalloc(&p_device, bytes));
    CHECK_CUDA_ERROR(hipMemcpy(p_device, p, bytes, hipMemcpyHostToDevice));
}

void cudaFreeMemRoutines(Body *p_device, float * buf){
    hipFree(p_device);
    free(buf); // Asegurarse de liberar la memoria de buf también

}

