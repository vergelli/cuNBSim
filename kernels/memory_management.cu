#include <hip/hip_runtime.h>
#include"body.cuh"


void bodyForceMalloc( int bytes, Body *p, Body *p_device){

    hipMalloc(&p_device,bytes);
    hipMemcpy(p_device, p, bytes, hipMemcpyHostToDevice);

}

void cudaFreeMemRoutines(Body *p_device){
    hipFree(p_device);
}

