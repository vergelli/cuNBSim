#include <stdio.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include "body.cuh"
#include "bodyForce.cuh"
#include "cuda_utils.cuh"

void bodyForceWraper( int nBodies, float dt, Body *p_device){

    int warpDim;
    int deviceId;
    int numberOfSMs;
    int gridDimX;
    int blockDimX;

    CHECK_CUDA_ERROR( 
        hipGetDevice(&deviceId));

    CHECK_CUDA_ERROR( 
        hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId));

    CHECK_CUDA_ERROR( 
        hipDeviceGetAttribute(&warpDim, hipDeviceAttributeWarpSize, deviceId));

    gridDimX = warpDim * numberOfSMs;

    blockDimX = warpDim * warpDim; 

    //*    stride = gridDimX * blockDimX;

    dim3 dimGrid(gridDimX, 1, 1);

    dim3 BodyForceDimBlock(blockDimX, 1, 1);

    // Llamada al kernel con los valores calculados
    bodyForceCUDA<<<dimGrid, BodyForceDimBlock>>>(p_device, dt, nBodies);

    CHECK_CUDA_ERROR(
        hipDeviceSynchronize());

};
