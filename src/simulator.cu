#include <hiprand/hiprand_kernel.h>
#include <iostream>
#include <cstdlib>
#include "body.cuh"
#include "utils.hpp"
#include "config.hpp"
#include "memory_management.cuh"
#include "deviceProps.cuh"
#include "kernelsInit.cu"
#include "boxMullerWraper.cuh"
#include "massWraper.cuh"
#include "velocityWrapper.cuh"
#include "bodyForceWraper.cuh"
#include "integrateWraper.cuh"
#include "data_collector.cuh"

int main(int argc, char* argv[]) {

    std::string config_path = (argc > 1) ? argv[1] : default_config_path;

    DeviceProperties deviceProps = getDeviceProps();
    load_config_from_file(config_path, deviceProps);

    int bytes = nBodies * sizeof(Body);
    float *buf;
    buf = (float *)malloc(bytes);
    Body *p = (Body*)buf;
    Body *p_device;
    hiprandState *d_states;

    allocateMemoryForParticles(bytes, p, p_device, d_states, nBodies);

    //~ Inicializacion de los parametros de lanzamiento de los kernels
    kernelsLaunchParamsInit(gridDimX, blockDimX, integrateStride, deviceProps);

    //~ Definiendo posicion inicial de las particulas
    execBoxMuller( nBodies, d_states, p_device, gridDimX, blockDimX);

    //~ Definiendo la masa inicial de las particulas
    massKernelLaunch( nBodies, p_device, gridDimX, blockDimX);

    //~ Definiendo la velocidad inicial de las particulas
    velocityKernelLaunch( nBodies, p_device, gridDimX, blockDimX, max_particles_speed);

    //~ Ciclo principal
    for (int iter = 0; iter < nIters; iter++) {
        execBodyForce(nBodies, dt, p_device, gridDimX, blockDimX);
        execIntegrate(nBodies, dt, p_device, gridDimX, blockDimX, integrateStride);
        simulationDataCollection(p, p_device, nBodies, bytes, iter);
        printProgress(iter + 1, nIters);
    }

    printf("INFO - Simulation terminated\n");

    //~ Rutinas de liberacion de memoria
    cudaFreeMemRoutines(p_device,d_states, buf);
    return 0;
}
