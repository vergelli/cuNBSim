#include <hip/hip_runtime.h>
#include "cuda_utils.cuh" // Asegúrate de tener esta cabecera para CHECK_CUDA_ERROR
#include "deviceProps.cuh"

// Función para obtener las propiedades del dispositivo
DeviceProperties getDeviceProps() {

    DeviceProperties props;

    CHECK_CUDA_ERROR(hipGetDevice(&props.deviceId));
    CHECK_CUDA_ERROR(hipDeviceGetAttribute(&props.numberOfSMs,
        hipDeviceAttributeMultiprocessorCount, 
        props.deviceId));
    CHECK_CUDA_ERROR(hipDeviceGetAttribute(&props.warpDim, 
        hipDeviceAttributeWarpSize, 
        props.deviceId));

    return props;
}
