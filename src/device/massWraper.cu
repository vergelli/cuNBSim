#include <stdio.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include "body.cuh"
#include "mass.cuh"
#include "cuda_utils.cuh"
#include "deviceProps.cuh"

void initMassKernelLaunch(
    int &gridDimX, 
    int &massDimX, 
    DeviceProperties deviceProps) {

    gridDimX = 2*deviceProps.warpDim;
    massDimX = deviceProps.warpDim * deviceProps.warpDim;
    printf("INFO - gridDimX: %d, massDimX: %d\n", gridDimX, massDimX);
}

void massKernelLaunch(
    int nBodies, 
    Body *p_device, 
    int gridDimX, 
    int blockDimX) {

    dim3 dimGrid(gridDimX, 1, 1);
    dim3 massDimBlock(blockDimX, 1, 1);
    initialize_mass<<<dimGrid, massDimBlock>>>(p_device, nBodies);
    CHECK_CUDA_ERROR(hipDeviceSynchronize());
}
