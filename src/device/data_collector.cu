#include <filesystem>  // Para crear directorios (C++17 en adelante)
#include <fstream>
#include <iostream>
#include <string>
#include "body.cuh"
#include "config.hpp"
#include "cuda_utils.cuh"
namespace fs = std::filesystem;

void logSimulationData(Body* p, int nBodies, int iter, std::string numerical_integrator) {

    //* Definir la ruta del archivo
    std::string file_path = std::string(data_directory) + simulation_data_file_name;

    //* Verificar si la carpeta 'data' existe, si no, crearla
    std::string data_directory_str = std::string(data_directory); 
    if (!fs::exists(data_directory_str)) {
        if (!fs::create_directories(data_directory_str)) {
            std::cerr << "ERROR: Could not create the directory: " << data_directory_str << std::endl;
            return;
        }
    }

    //* Verificar si el archivo ya existe
    bool fileExists = fs::exists(file_path);

    //* Abrir el archivo en modo append
    std::ofstream csvFile(file_path, std::ios_base::app);
    
    //* Validar si el archivo se abrió correctamente
    if (!csvFile.is_open()) {
        std::cerr << "ERROR: Can't open the file " << file_path << " to write data." << std::endl;
        return;
    }

    if (numerical_integrator == "euler-explicit") {
        if (!fileExists) {
            csvFile << "Iteration,BodyID,PosX,PosY,PosZ,VelX,VelY,VelZ,mass\n";
        }
        //* Escribir los datos de la simulación en el archivo
        for (int i = 0; i < nBodies; i++) {
            csvFile << iter << "," << i << "," << p[i].x << "," << p[i].y << "," << p[i].z << "," 
                    << p[i].vx << "," << p[i].vy << "," << p[i].vz << "," << p[i].mass << "\n";
        }
        /* code */
    } else if (numerical_integrator == "leap-frog") {
        if (!fileExists) {
            csvFile << "Iteration,BodyID,PosX,PosY,PosZ,mass\n";
        }
        //* Escribir los datos de la simulación en el archivo
        for (int i = 0; i < nBodies; i++) {
            csvFile << iter << "," << i << "," << p[i].x << "," << p[i].y << "," << p[i].z << "," 
                    "," << p[i].mass << "\n";
        }
    }

    //* Cerrar el archivo
    csvFile.close();

    //* Validar si el archivo se cerró correctamente
    if (csvFile.fail()) {
        std::cerr << "ERROR: File could not close properly." << std::endl;
    }
}

void simulationDataCollection(
    Body* p, 
    Body* p_device, 
    int nBodies, 
    int bytes, 
    int iter,
    std::string numerical_integrator) {

    CHECK_CUDA_ERROR(hipMemcpy(p, p_device, bytes, hipMemcpyDeviceToHost));
    logSimulationData(p, nBodies, iter, numerical_integrator);
}
