#include <stdio.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include "body.cuh"
#include "velocity.cuh"
#include "cuda_utils.cuh"
#include "deviceProps.cuh"

void initVelocityKernelLaunch(
    int &gridDimX, 
    int &VelocityDimX, 
    DeviceProperties deviceProps) {

    gridDimX = 2*deviceProps.warpDim;
    VelocityDimX = deviceProps.warpDim * deviceProps.warpDim;
    printf("INFO - gridDimX: %d, VelocityDimX: %d\n", gridDimX, VelocityDimX);
}

void velocityKernelLaunch(
    int nBodies, 
    Body *p_device, 
    int gridDimX, 
    int blockDimX,
    float max_particles_speed) {

    dim3 dimGrid(gridDimX, 1, 1);
    dim3 VelocityDimBlock(blockDimX, 1, 1);
    initialize_velocity<<<dimGrid, VelocityDimBlock>>>(p_device, nBodies, max_particles_speed);
    CHECK_CUDA_ERROR(hipDeviceSynchronize());
}
