#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include "body.cuh"
#include "integrate.cuh"
#include "cuda_utils.cuh"

void integrateWraper( int nBodies, float dt, Body *p_device){

    int warpDim;
    int deviceId;
    int numberOfSMs;
    int gridDimX;
    int blockDimX;
    int stride;

    CHECK_CUDA_ERROR( 
        hipGetDevice(&deviceId));

    CHECK_CUDA_ERROR( 
        hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId));

    CHECK_CUDA_ERROR( 
        hipDeviceGetAttribute(&warpDim, hipDeviceAttributeWarpSize, deviceId));


    gridDimX = warpDim*numberOfSMs;
    blockDimX =  sqrt(warpDim*warpDim); 
    stride = gridDimX * blockDimX;

    dim3 dimGrid(gridDimX, 1, 1);
    dim3 IntegrationDimBlock(blockDimX, 1, 1);

    integrateCUDA<<<dimGrid, IntegrationDimBlock>>>(p_device, dt, nBodies, stride);

    CHECK_CUDA_ERROR(
        hipDeviceSynchronize());

};
